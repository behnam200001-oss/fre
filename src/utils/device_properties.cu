#include "device_properties.h"
#include "../utils/logger.h"

namespace bitcoin_miner {

int DeviceProperties::get_device_count() {
    int count = 0;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        Logger::error("Failed to get device count: {}", hipGetErrorString(err));
        return 0;
    }
    return count;
}

DeviceProperties::DeviceInfo DeviceProperties::get_device_properties(int device_id) {
    DeviceInfo info;
    hipDeviceProp_t props;
    
    hipError_t err = hipGetDeviceProperties(&props, device_id);
    if (err != hipSuccess) {
        Logger::error("Failed to get device properties for device {}: {}", 
                     device_id, hipGetErrorString(err));
        return info;
    }
    
    info.name = props.name;
    info.totalGlobalMem = props.totalGlobalMem;
    info.multiProcessorCount = props.multiProcessorCount;
    info.maxThreadsPerBlock = props.maxThreadsPerBlock;
    info.major = props.major;
    info.minor = props.minor;
    
    return info;
}

} // namespace bitcoin_miner